#include "hip/hip_runtime.h"
#include "Cuda/lib.cu.h"

//Cuda libraries includes
#include <hip/hip_runtime.h>

//Local
#include "Cuda/helper.h"

//Kernel definition
__global__ void printKernel() {
  printf("This is libCall from LibObj from Cuda");
}

void libCallCuda() {
  PUSH_NVCTX("MyGPUPrint",0)
  printKernel<<<1,1,0,0>>>();
  checkCudaErrors(hipDeviceSynchronize());
  POP_NVCTX
}
